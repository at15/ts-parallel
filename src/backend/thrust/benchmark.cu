#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../../benchmark.hpp"

namespace aya
{
namespace bench
{
void ping()
{
    std::cout << "pong from thrust benchmark backend" << std::endl;
}

std::string name()
{
    return "thrust";
}

template <typename T>
struct ThrustBenchmarkBackend : BenchmarkBackend<T>
{
    ~ThrustBenchmarkBackend()
    {
        std::cout << "thrust benchmark backend destructor called" << std::endl;
    }

    void generate(int num)
    {
        std::cout << "generateing !\n";
        h_vec.resize(num);
        std::generate(h_vec.begin(), h_vec.end(), rand);
        std::cout << "generated !\n";
    }

    void copy()
    {
        d_vec = h_vec;
        hipDeviceSynchronize();
    }

    void sort()
    {
        std::cout << "start sorting\n";
        thrust::sort(d_vec.begin(), d_vec.end());
        hipDeviceSynchronize();
        std::cout << "sort finished\n";
    }

    void reduce()
    {
        thrust::reduce(d_vec.begin(), d_vec.end());
        hipDeviceSynchronize();
    }

  private:
    thrust::host_vector<T> h_vec;
    thrust::device_vector<T> d_vec;
};

template <typename T>
BenchmarkBackend<T> *init()
{
    return new ThrustBenchmarkBackend<T>;
}

} // bench
} // aya::bench

int main(int argc, char **argv)
{
    return aya::bench::launch(argc, argv);
}