#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../../benchmark.hpp"

namespace aya
{
namespace bench
{
void ping()
{
    std::cout << "pong from thrust benchmark backend" << std::endl;
}

std::string name()
{
    return "thrust";
}

template <typename T>
void sort(int num)
{
    std::cout << "sort " << num << std::endl;

    // FIXME: static cause the following problem
    //     terminate called after throwing an instance of 'thrust::system::system_error'
    //   what():  hipFree in free: driver shutting down
    // static thrust::device_vector<T> d_vec;
    thrust::device_vector<T> d_vec;

    thrust::host_vector<T> h_vec(num);
    std::generate(h_vec.begin(), h_vec.end(), rand);
    d_vec = h_vec;
    hipDeviceSynchronize();
    thrust::sort(d_vec.begin(), d_vec.end());
    hipDeviceSynchronize();

    std::cout << "sort finished" << std::endl;
}

template <typename T>
struct ThrustBenchmarkBackend : BenchmarkBackend<T>
{
    ~ThrustBenchmarkBackend()
    {
        std::cout << "thrust benchmark backend destructor called" << std::endl;
    }

    void generate(int num)
    {
        std::cout << "generateing !\n";
        h_vec.resize(num);
        std::generate(h_vec.begin(), h_vec.end(), rand);
        std::cout << "generated !\n";
    }

    void copy()
    {
        d_vec = h_vec;
        hipDeviceSynchronize();
    }

    void sort()
    {
        std::cout << "start sorting\n";
        thrust::sort(d_vec.begin(), d_vec.end());
        hipDeviceSynchronize();
        std::cout << "sort finished\n";
    }

  private:
    thrust::host_vector<T> h_vec;
    thrust::device_vector<T> d_vec;
};

template <typename T>
BenchmarkBackend<T> *init()
{
    return new ThrustBenchmarkBackend<T>;
}
}
}

int main(int argc, char **argv)
{
    return aya::bench::launch(argc, argv);
}