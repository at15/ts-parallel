#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../../benchmark.hpp"

namespace aya
{
namespace bench
{
void ping()
{
    std::cout << "pong from thrust benchmark backend" << std::endl;
}

std::string name()
{
    return "thrust";
}

template<typename T>
void sort(int num)
{
    std::cout << "sort " << num << std::endl;

// FIXME: static cause the following problem    
//     terminate called after throwing an instance of 'thrust::system::system_error'
//   what():  hipFree in free: driver shutting down
    // static thrust::device_vector<T> d_vec;
    thrust::device_vector<T> d_vec;
    
    thrust::host_vector<T> h_vec(num);
    std::generate(h_vec.begin(), h_vec.end(), rand);
    d_vec = h_vec;
    hipDeviceSynchronize();
    thrust::sort(d_vec.begin(), d_vec.end());
    hipDeviceSynchronize();

    std::cout << "sort finished" << std::endl;
}
}
}

int main(int argc, char **argv)
{
   return aya::bench::launch(argc, argv);
}