
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <math.h>
#include <algorithm>

using namespace std;

// nvcc -ccbin clang++-3.8 cufp16_test.cu
// NOTE: didn't get any error like building ArrayFire
int main()
{
   cout << isinf(NAN) << endl; // 0
   cout << isinf(INFINITY) << endl; // 1
}
