
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <cmath>
#include <cfloat>

using namespace std;

// nvcc -ccbin clang++-3.8 cufp16_test.cu
// NOTE: didn't get any error like building ArrayFire
int main()
{
   cout << isinf(NAN) << endl; // 0
   cout << isinf(INFINITY) << endl; // 1
}
